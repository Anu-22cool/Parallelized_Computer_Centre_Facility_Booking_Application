#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>

#define max_N 100000
#define max_P 30
#define BLOCKSIZE 1024

using namespace std;

//*******************************************

// Write down the kernels here

// First I am calculating the prefix sum of facids capacity so that i can map centre no and facility ids of the processing request.
// Here i am calculating prefix sum using Reduction method.
// After that I am sending threads which is equal to total no of facility.
// Each thread represing each facility.
// After that I have made worklist  to push all the request which are going to use that facility and slot array  of 25 hours which is intially zero.
// Then I will check whether the facility his free for the that request if it is allowed it will process further otherwise it wont update anything.

__global__ void  exclusivesum(int n, int *garr , int* ccopy,int *csum)
{
    
    if(threadIdx.x+ blockDim.x*blockIdx.x<n){
     garr[threadIdx.x+ blockDim.x*blockIdx.x]-=ccopy[threadIdx.x+ blockDim.x*blockIdx.x];
     if((threadIdx.x+ blockDim.x*blockIdx.x) == n-1)
      csum[0] = ccopy[n-1] + garr[n-1];

    }
}


__global__ void  dkernel(int n,int *a,int start, int end)
{     
     int id=start+threadIdx.x;
  
     if(id<n)
     {
          int  tmp,off=1;
          while(off<n)
        {
            if(threadIdx.x>=off){ 
                tmp=a[id-off];
            __syncthreads();
              a[id]+=tmp;
            __syncthreads();
            }
            int d= off*2;
            off=d;
        } 
    }
}


__global__ void  dke(int L , int K , int C,int *sum, int *gcentre,int *gfacility,int *gcapacity,int *gfac_ids,int *gsucc_reqs,int *gtot_reqs,int *greq_id,int *greq_cen,int *greq_fac,int *greq_start,int *greq_slots,int *gfacps,int R,int N)
{
    int iterations=sum[0], id=threadIdx.x + blockIdx.x * blockDim.x;
    
    if(id < iterations)
    {

    int  counter  = 0;
    int capacity= gcapacity[id];
    int slot[25];
    int centerno=-1;
    int facilityno=-1; 
    int worklist[10000];
    
   
    int ii=0;
    while(ii<25)
    {
      slot[ii] = 0;
      ii++;
    }
    
    int i=0;
    while(i<N)
    { 
      int val=gfacps[i];
      if(id < val)
        {
          centerno = i - 1;
          facilityno =  id - gfacps[i-1];
          break;
        }
        i++;
    }

    if(centerno==-1)
     {
        centerno=N-1;
        facilityno = threadIdx.x + blockIdx.x * blockDim.x - gfacps[centerno];
     }

    int j=0;
    while(j<R)
    {
      if(greq_cen[j] == centerno )
      { 
         if( greq_fac[j] == facilityno){
          worklist[counter] = greq_id[j];
          counter++;
         }
      }
      j++;
    }
    
    
    for(int i=0;i<counter;i++)
    {

        int req=worklist[i], start_slot= greq_start[req], slot_count= greq_slots[req], check=0;
        for(int i=start_slot;i<start_slot+slot_count;i++)
        {
           if(slot[i]>=capacity)
            { 
              check=1;
              break;
            } 
        }

        if(check==0)
           for(int i=start_slot;i<start_slot+slot_count;i++)
             slot[i]=slot[i]+1;

        if(check==0)
           atomicAdd(&gsucc_reqs[centerno],1);
        
    }

    }

    
}
__global__ void  exclusivesum(int n, int *garr , int* ccopy,int *csum)
{
    
    if(threadIdx.x+ blockDim.x*blockIdx.x<n){
     garr[threadIdx.x+ blockDim.x*blockIdx.x]-=ccopy[threadIdx.x+ blockDim.x*blockIdx.x];
     if((threadIdx.x+ blockDim.x*blockIdx.x) == n-1)
      csum[0] = ccopy[n-1] + garr[n-1];

    }
}


__global__ void  dkernel(int n,int *a,int start, int end)
{     
     int id=start+threadIdx.x;
  
     if(id<n)
     {
          int  tmp,off=1;
          while(off<n)
        {
            if(threadIdx.x>=off){ 
                tmp=a[id-off];
            __syncthreads();
              a[id]+=tmp;
            __syncthreads();
            }
            int d= off*2;
            off=d;
        } 
    }
}





//***********************************************


int main(int argc,char **argv)
{
	// variable declarations...
    int N,*centre,*facility,*capacity,*fac_ids, *succ_reqs, *tot_reqs;
    

    FILE *inputfilepointer;
    
    //File Opening for read
    char *inputfilename = argv[1];
    inputfilepointer    = fopen( inputfilename , "r");

    if ( inputfilepointer == NULL )  {
        printf( "input.txt file failed to open." );
        return 0; 
    }

    fscanf( inputfilepointer, "%d", &N ); // N is number of centres
	
    // Allocate memory on cpu
    centre=(int*)malloc(N * sizeof (int));  // Computer  centre numbers
    facility=(int*)malloc(N * sizeof (int));  // Number of facilities in each computer centre
    fac_ids=(int*)malloc(max_P * N  * sizeof (int));  // Facility room numbers of each computer centre
    capacity=(int*)malloc(max_P * N * sizeof (int));  // stores capacities of each facility for every computer centre 


    int success=0;  // total successful requests
    int fail = 0;   // total failed requests
    tot_reqs = (int *)malloc(N*sizeof(int));   // total requests for each centre
    succ_reqs = (int *)malloc(N*sizeof(int)); // total successful requests for each centre

    // Input the computer centres data
    int k1=0 , k2 = 0;
    for(int i=0;i<N;i++)
    {
      fscanf( inputfilepointer, "%d", &centre[i] );
      fscanf( inputfilepointer, "%d", &facility[i] );
      
      for(int j=0;j<facility[i];j++)
      {
        fscanf( inputfilepointer, "%d", &fac_ids[k1] );
        k1++;
      }
      for(int j=0;j<facility[i];j++)
      {
        fscanf( inputfilepointer, "%d", &capacity[k2]);
        k2++;     
      }
    }

    // variable declarations
    int *req_id, *req_cen, *req_fac, *req_start, *req_slots;   // Number of slots requested for every request
    
    // Allocate memory on CPU 
	int R;
	fscanf( inputfilepointer, "%d", &R); // Total requests
    req_id = (int *) malloc ( (R) * sizeof (int) );  // Request ids
    req_cen = (int *) malloc ( (R) * sizeof (int) );  // Requested computer centre
    req_fac = (int *) malloc ( (R) * sizeof (int) );  // Requested facility
    req_start = (int *) malloc ( (R) * sizeof (int) );  // Start slot of every request
    req_slots = (int *) malloc ( (R) * sizeof (int) );   // Number of slots requested for every request
    
    // Input the user request data
    for(int j = 0; j < R; j++)
    {
       fscanf( inputfilepointer, "%d", &req_id[j]);
       fscanf( inputfilepointer, "%d", &req_cen[j]);
       fscanf( inputfilepointer, "%d", &req_fac[j]);
       fscanf( inputfilepointer, "%d", &req_start[j]);
       fscanf( inputfilepointer, "%d", &req_slots[j]);
       tot_reqs[req_cen[j]]+=1;  
    }
		


    //*********************************
    // Call the kernels here
    int *gcentre,*gfacility,*gcapacity,*gfac_ids, *gsucc_reqs, *gtot_reqs,*greq_id, *greq_cen, *greq_fac, *greq_start, *greq_slots;
    hipMalloc(&gcapacity,sizeof(int)*max_P * N );
    int *csum;
    hipMalloc(&csum,sizeof(int));
    hipMalloc(&gfac_ids,sizeof(int)*max_P * N );
    hipMalloc(&gsucc_reqs,sizeof(int)*N);
    hipMalloc(&gtot_reqs,sizeof(int)*N);
    hipMalloc(&greq_id,sizeof(int)*R);
    hipMalloc(&greq_cen,sizeof(int)*R);
    hipMalloc(&greq_fac,sizeof(int)*R);
    hipMalloc(&greq_start,sizeof(int)*R);
    hipMalloc(&greq_slots,sizeof(int)*R);
    hipMalloc(&gcentre,sizeof(int)*N);
    hipMalloc(&gfacility,sizeof(int)*N);
   
    
   
    
    
    hipMemcpy(greq_id, req_id, R * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(greq_cen, req_cen, R * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(greq_fac, req_fac, R * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(greq_start, req_start, R * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(greq_slots, req_slots, R * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gcentre,centre , N* sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gfacility, facility, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gcapacity,capacity , max_P * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gfac_ids, fac_ids, max_P * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gsucc_reqs,succ_reqs ,  N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gtot_reqs,tot_reqs, N * sizeof(int), hipMemcpyHostToDevice);

    hipMemset(gsucc_reqs, 0, N * sizeof(int));
    int *gfacps,*ccopy;
    hipMalloc(&gfacps,sizeof(int)*(N));
    hipMalloc(&ccopy,sizeof(int)*(N));
    hipMemcpy(gfacps,facility, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(ccopy,facility, N * sizeof(int), hipMemcpyHostToDevice);
    
    int blocksize=1023;
    int noofblocks= ceil((float)N/blocksize);
    

    for(int i=0;i<noofblocks;i++)
    {  
        int start=i*blocksize, end;
        if(i!=noofblocks-1)
           end= (i+1)*blocksize-1;
        else
            end=N-1;

       
       if(i==0)
       dkernel<<<1,blocksize>>>(N,gfacps,start,end);
       else
       dkernel<<<1,blocksize+1>>>(N,gfacps,start-1,end);
       hipDeviceSynchronize();
    }

    exclusivesum<<<noofblocks,1024>>>(N,gfacps,ccopy,csum);
    int sum[1];
    hipMemcpy(sum,csum,sizeof(int),hipMemcpyDeviceToHost);
    int threadsPerBlock = 1024;
    int numBlocks = (sum[0] + threadsPerBlock - 1) / threadsPerBlock;
    
    dke<<<numBlocks, threadsPerBlock>>>(0,0,0,csum, gcentre, gfacility, gcapacity, gfac_ids, gsucc_reqs, gtot_reqs, greq_id, greq_cen, greq_fac, greq_start, greq_slots, gfacps, R, N);
    hipDeviceSynchronize();

    hipMemcpy(succ_reqs ,gsucc_reqs,  N * sizeof(int), hipMemcpyDeviceToHost);
    for(int i=0;i<N;i++)
     success+=succ_reqs[i];

    fail=R-success; 
    
     

  


    //********************************





    // Output
    char *outputfilename = argv[2]; 
    FILE *outputfilepointer;
    outputfilepointer = fopen(outputfilename,"w");

    fprintf( outputfilepointer, "%d %d\n", success, fail);
    //printf("%d %d\n", success, fail);
    for(int j = 0; j < N; j++)
    {
        fprintf( outputfilepointer, "%d %d\n", succ_reqs[j], tot_reqs[j]-succ_reqs[j]);
        //printf("%d %d\n", succ_reqs[j], tot_reqs[j]-succ_reqs[j]);
    }
    fclose( inputfilepointer );
    fclose( outputfilepointer );
    hipDeviceSynchronize();
	return 0;
}
